#include <cassert>
#include <cstdio>
#include <cstdlib>
#include "grasta_cuda_util.cuh"
#include "grasta_reduction.cuh"

using namespace std;

hipError_t cudaSimpleReduction(float* data_to_sum, unsigned int num, float &accu){
    float *dev_data_to_sum = 0; // array of elements to sum that reside on the device
    float *dev_temp_sums = 0;   // holds accumulations of elements between kernel calls
    float *temp_sums = 0;       // storage for temps sums from device
    hipError_t cudaStatus;

    temp_sums = (float*) malloc(REDUCTION_BLOCK_SIZE * sizeof(float));

    // This invariant makes the reduction algorithm easier to implement
    assert(num % (REDUCTION_BLOCK_SIZE * 2) == 0);

    // Allocate GPU buffers for element
    cudaStatus = hipMalloc((void**)&dev_data_to_sum, num * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_temp_sums, REDUCTION_BLOCK_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_data_to_sum, data_to_sum, num * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU
    const unsigned int kSMEM_BYTES = REDUCTION_BLOCK_SIZE * sizeof(float);
    simple_reduce<REDUCTION_BLOCK_SIZE><<< REDUCTION_NUM_BLOCKS, REDUCTION_BLOCK_SIZE, kSMEM_BYTES >>>
        (dev_data_to_sum, dev_temp_sums, num / (REDUCTION_BLOCK_SIZE * 2)); // kernel args

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(temp_sums, dev_temp_sums, REDUCTION_BLOCK_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_temp_sums);
    hipFree(dev_data_to_sum);
    free(temp_sums);
    
    return cudaStatus;
}
