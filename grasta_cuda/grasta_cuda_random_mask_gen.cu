#include "hip/hip_runtime.h"
#include "grasta_cuda_random_mask_gen.cuh"
#include "grasta_reduction.cuh"
#include "grasta_cuda_util.cuh"
#include <cmath>
#include <algorithm>
#include <ctime>
#include <iostream>

// HELPER FUNCTION DECLARATIONS
///////////////////////////////
bool MaskAlreadyContainsIndex(int *const maskBase, 
                              const int maskSize, 
                              const int value);

/* Default constructor
    m_maskSize is a multiple of REDUCTION_BLOCKSIZE_SIZE rounded up. This allows
    for performance improvements in some CUDA kernels.
*/
RandomMaskGenerator::RandomMaskGenerator()
    : p_data(nullptr),
      m_maskSize( (((int)((float)(kSCREEN_HEIGHT * kSCREEN_HEIGHT) * kRANDOMSAMPLEPERCENTAGE)) + REDUCTION_BLOCK_SIZE)
                 - ((int)((float)(kSCREEN_HEIGHT * kSCREEN_HEIGHT) * kRANDOMSAMPLEPERCENTAGE)) % REDUCTION_BLOCK_SIZE),
      m_gen(std::clock()),
      m_distribution(0, kSCREEN_HEIGHT * kSCREEN_WIDTH)
{
    
    // Allocate memory for all the sample masks and initialize all bits to 1
    p_data = new int[m_maskSize * kNUMRANDOMMASKS];
    std::fill_n(p_data, m_maskSize * kNUMRANDOMMASKS, ULONG_MAX);

    std::cout << "Mask Size: " << m_maskSize << std::endl;

    for(unsigned int i = 0; i < kNUMRANDOMMASKS; ++i)
    {
        std::cout << "Making Mask " << i << std::endl;

        for(int j = 0; j < m_maskSize; ++j)
        {
            int newIndex = m_distribution(m_gen);
            while( MaskAlreadyContainsIndex(p_data + (i * m_maskSize), m_maskSize, newIndex))
            {
                newIndex = m_distribution(m_gen);
            }
            p_data[i * m_maskSize + j] = newIndex;
        }
    }
}

RandomMaskGenerator::~RandomMaskGenerator()
{
    delete [] p_data;
    p_data = nullptr;
    p_Instance = nullptr;
}

int* RandomMaskGenerator::GetRandomMask()
{
    return p_data + (m_gen() % kNUMRANDOMMASKS) * m_maskSize;
}

const int RandomMaskGenerator::GetMaskSize()
{
    return m_maskSize;
}

RandomMaskGenerator* RandomMaskGenerator::p_Instance = nullptr;

RandomMaskGenerator* RandomMaskGenerator::Instance()
{
    if(p_Instance == nullptr)
    {
        p_Instance = new RandomMaskGenerator();
    }
    return p_Instance;
}

bool MaskAlreadyContainsIndex(int * const maskBase, 
                              const int maskSize, 
                              const int value)
{
    for(int i = 0; i < maskSize; ++i)
    {
        if(maskBase[i] == value)
        {
            return true;
        }
    }
    return false;
}
